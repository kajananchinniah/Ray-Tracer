#include "common/check_cuda_errors.hpp"
#include "common/cuda_memory_utils.hpp"

namespace RayTracer
{
namespace cuda
{
namespace lowlevel
{
void *allocateCudaMemory(u64 byte_size)
{
    void *ptr;
    CHECK_CUDA_ERRORS(hipMallocManaged(&ptr, byte_size));
    return ptr;
}

void prefetchToGpu(const void *ptr, u64 byte_size)
{
    int device{-1};
    hipGetDevice(&device);
    CHECK_CUDA_ERRORS(hipMemPrefetchAsync(ptr, byte_size, device, NULL));
}

void prefetchToCpu(const void *ptr, u64 byte_size)
{
    CHECK_CUDA_ERRORS(hipMemPrefetchAsync(ptr, byte_size, hipCpuDeviceId));
}

void copyCudaMemory(void *dst, const void *src, u64 byte_size,
                    hipMemcpyKind kind)
{
    CHECK_CUDA_ERRORS(hipMemcpy(dst, src, byte_size, kind));
}

void deallocateCudaMemory(void *ptr)
{
    CHECK_CUDA_ERRORS(hipFree(ptr));
}

} // namespace lowlevel

void waitForCuda()
{
    CHECK_CUDA_ERRORS(hipGetLastError());
    CHECK_CUDA_ERRORS(hipDeviceSynchronize());
}
} // namespace cuda

} // namespace RayTracer
